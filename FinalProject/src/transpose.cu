#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "include/import_sparse_matrix.h"

using namespace std;

#define NUM_REPS 10



__global__ void warm_up_gpu(){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + tid; 
}


void transpose_cuSparse_CSR(string file){
    // load CSR matrix from file
    int rows, columns, nnz;
    int *row_offsets, *col_indices;
    float* values;

    csr_from_file(file, rows, columns, nnz, row_offsets, col_indices, values);

    // create CSR matrix using cuSparse
    hipsparseSpMatDescr_t sparse_matrix;

    // void* for the three arrays
    int *dev_row_offsets, *dev_col_indices;
    float* dev_values;

    // allocate memory on device
    hipMalloc(&dev_row_offsets, (rows+1) * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));
    hipMalloc(&dev_values,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_offsets, row_offsets, (rows+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);
    
    // create CSR matrix
    hipsparseCreateCsr(&sparse_matrix, rows, columns, nnz, dev_row_offsets, dev_col_indices, dev_values, HIP_R_32I, 
                        HIP_R_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    // transpose
    
    // destroy matrix
    hipsparseDestroySpMat(sparse_matrix);

    // free device memory
    hipFree(dev_rows_offsets);
    hipFree(dev_col_indices);
    hipFree(dev_values);
}


void transpose_cuSparse_COO(string file){
    // load COO matrix from file
    int rows, cols, nnz;
    int *row_indices, *col_indices;
    float* values;

    coo_from_file(file, rows, cols, nnz, row_indices, col_indices, values);

    // create COO CUDA matrix using cuSparse 
    hipsparseSpMatDescr_t sparse_matrix;

    // void* for the three arrays
    int *dev_row_indices, *dev_col_indices;
    float* dev_values;

    // allocate memory on device
    hipMalloc(&dev_row_indices, nnz * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));
    hipMalloc(&dev_values,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_indices, row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);

    // create COO matrix
    hipsparseCreateCoo(&sparse_matrix, rows, columns, nnz, dev_row_indices, dev_col_indices, dev_values, HIP_R_32I, 
                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    // transpose 

    // copy back


    // destroy matrix
    hipsparseDestroySpMat(sparse_matrix);

    // free device memory
    hipFree(dev_rows_indices);
    hipFree(dev_col_indices);
    hipFree(dev_values);
}


int main(int argc, char* argv[]){
    transpose_cuSparse_COO("test_matrices/coo/1-bp_200_coo.csv")
    
    return 0;
}

