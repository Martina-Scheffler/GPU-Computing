#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "../include/import_sparse_matrix.h"

using namespace std;

#define NUM_REPS 10



__global__ void warm_up_gpu(){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + tid; 
}


void transpose_cuSparse_CSR(string file, string timing_file){
    // file to save execution time for bandwidth analysis
    std::ofstream myfile;
	myfile.open(timing_file);

    // load CSR matrix from file
    int rows, columns, nnz;
    int *row_offsets, *col_indices;
    float* values;

    csr_from_file(file, rows, columns, nnz, row_offsets, col_indices, values);

    // create arrays on device
    int *dev_row_offsets, *dev_col_indices;
    float* dev_values;

    // allocate memory on device
    hipMalloc(&dev_row_offsets, (rows+1) * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));
    hipMalloc(&dev_values,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_offsets, row_offsets, (rows+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);

    // Create CUDA events to use for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // reserve buffer space necessary for the transpose
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    int *dev_tp_row_indices, *dev_tp_col_offsets;
    float* dev_tp_values;
    hipMalloc(&dev_tp_row_indices, nnz * sizeof(int));
    hipMalloc(&dev_tp_col_offsets, (columns+1) * sizeof(int));
    hipMalloc(&dev_tp_values, nnz * sizeof(float));

    size_t buffer_size;
    hipsparseCsr2cscEx2_bufferSize(handle, rows, columns, nnz, dev_values, dev_row_offsets, dev_col_indices, 
                                    dev_tp_values, dev_tp_col_offsets, dev_tp_row_indices, HIP_R_32F, 
                                    HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
                                    &buffer_size); 

    void* buffer;
    hipMalloc(&buffer, buffer_size);

    // start CUDA timer 
    hipEventRecord(start, 0);

    // run NUM_REPS times
    for (int i=0; i<NUM_REPS; i++){
        // transpose by converting from CSR to CSC
        hipsparseCsr2cscEx2(handle, rows, columns, nnz, dev_values, dev_row_offsets, dev_col_indices, dev_tp_values, 
                            dev_tp_col_offsets, dev_tp_row_indices, HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, 
                            HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, buffer);
    }

    // synchronize - TODO: necessary?
    hipDeviceSynchronize();

    // stop CUDA timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // divide by NUM_REPS to get mean
    milliseconds /= NUM_REPS;

    // save execution time and buffer size to file
    myfile << milliseconds << "\n";
    myfile << rows << "\n";
    myfile << columns << "\n";
    myfile << nnz << "\n";
    myfile << buffer_size << "\n";

    // copy results back to host
    int *row_offsets_tp = (int*) malloc((columns+1) * sizeof(int));
    int *col_indices_tp = (int*) malloc(nnz * sizeof(int));
    float* values_tp = (float*) malloc(nnz * sizeof(float));

    hipMemcpy(col_indices_tp, dev_tp_row_indices, nnz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(row_offsets_tp, dev_tp_col_offsets, (columns + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(values_tp, dev_tp_values, nnz * sizeof(float), hipMemcpyDeviceToHost);

    // save transposed matrix to file
    transposed_csr_to_file(file, columns, rows, nnz, row_offsets_tp, col_indices_tp, values_tp);

    // close file
	myfile.close();

    // free timer events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // destroy handle
    hipsparseDestroy(handle);

    // free device memory
    hipFree(dev_row_offsets);
    hipFree(dev_col_indices);
    hipFree(dev_values);

    hipFree(dev_tp_row_indices);
    hipFree(dev_tp_col_offsets);
    hipFree(dev_tp_values);

    // free host memory
    free(row_offsets);
    free(col_indices);
    free(values);
    free(row_offsets_tp);
    free(col_indices_tp);
    free(values_tp);
}


void transpose_cuSparse_COO(string file){
    // load COO matrix from file
    int rows, columns, nnz;
    int *row_indices, *col_indices;
    float* values;

    coo_from_file(file, rows, columns, nnz, row_indices, col_indices, values);

    // create COO CUDA matrix using cuSparse 
    hipsparseSpMatDescr_t sparse_matrix;

    // void* for the three arrays
    int *dev_row_indices, *dev_col_indices;
    float* dev_values;

    // allocate memory on device
    hipMalloc(&dev_row_indices, nnz * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));
    hipMalloc(&dev_values,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_indices, row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);

    // create COO matrix
    hipsparseCreateCoo(&sparse_matrix, rows, columns, nnz, dev_row_indices, dev_col_indices, dev_values, HIPSPARSE_INDEX_32I, 
                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    // reserve necessary buffer space
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);
    const int alpha = 1;
    const int beta = 0;
    hipsparseConstDnVecDescr_t vector = NULL;
    size_t buffer_size;

    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_TRANSPOSE, alpha, sparse_matrix, vector, beta, vector, 
                            HIP_R_32F, HIPSPARSE_SPMV_COO_ALG1, &buffer_size);

    void* buffer;
    hipMalloc(&buffer, buffer_size);
                        
    // preprocess


    // transpose 

    // copy back


    // destroy matrix
    hipsparseDestroySpMat(sparse_matrix);

    // destroy handle
    hipsparseDestroy(handle);

    // free device memory
    hipFree(dev_row_indices);
    hipFree(dev_col_indices);
    hipFree(dev_values);

    // free host memory 
    free(row_indices);
    free(col_indices);
    free(values);
}


int main(int argc, char* argv[]){
    if (argc < 2){
        throw runtime_error("Please choose a strategy");
    }

    // Strategy 0: cuSPARSE CSR
    if (atoi(argv[1]) == 0){
        printf("Use CSR format and the cuSPARSE library.\n");

        // check which test matrix to use
        if (argc < 3){
            throw runtime_error("Please choose a test matrix");
        }

        string argv2 = argv[2];
        if (argv2 == "all"){
            for (int i=1; i<11; i++){
                printf("Transposing matrix %d\n", i);
                transpose_cuSparse_CSR("test_matrices/csr/" + to_string(i) + "_csr.csv", 
                                        "output/csr_cusparse_" + to_string(i) + ".csv");
            }
        }
        else {
            printf("Transposing matrix %d\n", atoi(argv[2]));
            transpose_cuSparse_CSR("test_matrices/csr/" + to_string(atoi(argv[2])) + "_csr.csv",
                                    "output/csr_cusparse_" + to_string(atoi(argv[2])) + ".csv");
        }
    }

    // Strategy 1: cuSPARSE COO
    if (atoi(argv[1]) == 1){
        printf("Use COO format and the cuSPARSE library.\n");

        // check which test matrix to use
        if (argc < 3){
            throw runtime_error("Please choose a test matrix");
        }

        string argv2 = argv[2];
        if (argv2 == "all"){
            for (int i=1; i<11; i++){
                printf("Transposing matrix %d\n", i);
                transpose_cuSparse_COO("test_matrices/coo/" + to_string(i) + "_coo.csv");
            }
        }
        else {
            printf("Transposing matrix %d\n", atoi(argv[2]));
            transpose_cuSparse_COO("test_matrices/coo/" + to_string(atoi(argv[2])) + "_coo.csv");
        }
    }
    
    return 0;
}

