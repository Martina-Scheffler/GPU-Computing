#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "../include/import_sparse_matrix.h"

using namespace std;

#define NUM_REPS 10



__global__ void warm_up_gpu(){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + tid; 
}


void transpose_cuSparse_CSR(string file){
    // load CSR matrix from file
    int rows, columns, nnz;
    int *row_offsets, *col_indices;
    float* values;

    csr_from_file(file, rows, columns, nnz, row_offsets, col_indices, values);

    // for (int i=0; i<rows+1; i++){
    //     printf("%d\n", row_offsets[i]);
    // }

    // // create CSR matrix using cuSparse
    // hipsparseSpMatDescr_t sparse_matrix;

    // create arrays on device
    int *dev_row_offsets, *dev_col_indices;
    float* dev_values;

    // allocate memory on device
    hipMalloc(&dev_row_offsets, (rows+1) * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));
    hipMalloc(&dev_values,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_offsets, row_offsets, (rows+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);
    
    // create CSR matrix
    // hipsparseCreateCsr(&sparse_matrix, rows, columns, nnz, dev_row_offsets, dev_col_indices, dev_values, 
    //                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    // reserve buffer space necessary for the transpose
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    int *dev_tp_row_indices, *dev_tp_col_offsets;
    float* dev_tp_values;
    hipMalloc(&dev_tp_row_indices, nnz * sizeof(int));
    hipMalloc(&dev_tp_col_offsets, (columns+1) * sizeof(int));
    hipMalloc(&dev_tp_values, nnz * sizeof(float));

    size_t buffer_size;

    hipsparseCsr2cscEx2_bufferSize(handle, rows, columns, nnz, dev_values, dev_row_offsets, dev_col_indices, 
                                    dev_tp_values, dev_tp_col_offsets, dev_tp_row_indices, HIP_R_32F, 
                                    HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
                                    &buffer_size); 

    cout << buffer_size << "\n"; 
                              
    // transpose by converting from CSR to CSC
    void* buffer;
    hipMalloc(&buffer, buffer_size);
    hipsparseStatus_t result = hipsparseCsr2cscEx2(handle, rows, columns, nnz, dev_values, dev_row_offsets, dev_col_indices, dev_tp_values, 
                        dev_tp_col_offsets, dev_tp_row_indices, HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, 
                        HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, buffer);

    cout << result << "\n";

    // copy results back to host
    int *row_offsets_tp = (int*) malloc((columns+1) * sizeof(int));
    int *col_indices_tp = (int*) malloc(nnz * sizeof(int));
    float* values_tp = (float*) malloc(nnz * sizeof(float));

    hipMemcpy(col_indices_tp, dev_tp_row_indices, nnz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(row_offsets_tp, dev_tp_col_offsets, (columns + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(values_tp, dev_tp_values, nnz * sizeof(float), hipMemcpyDeviceToHost);

    for (int i=0; i<columns+1; i++){
        printf("%d\n", row_offsets_tp[i]);
    }

    // save transposed matrix to file
    transposed_csr_to_file(file, columns, rows, nnz, row_offsets_tp, col_indices_tp, values_tp);
    
    // // destroy matrix
    // hipsparseDestroySpMat(sparse_matrix);

    // destroy handle
    hipsparseDestroy(handle);

    // free device memory
    hipFree(dev_row_offsets);
    hipFree(dev_col_indices);
    hipFree(dev_values);

    hipFree(dev_tp_row_indices);
    hipFree(dev_tp_col_offsets);
    hipFree(dev_tp_values);

    // free host memory
    free(row_offsets);
    free(col_indices);
    free(values);
    free(row_offsets_tp);
    free(col_indices_tp);
    free(values_tp);
}


void transpose_cuSparse_COO(string file){
    cout << "Running cuSparse COO transpose" << "\n";
    // load COO matrix from file
    int rows, columns, nnz;
    int *row_indices, *col_indices;
    float* values;

    coo_from_file(file, rows, columns, nnz, row_indices, col_indices, values);

    // create COO CUDA matrix using cuSparse 
    hipsparseSpMatDescr_t sparse_matrix;

    // void* for the three arrays
    int *dev_row_indices, *dev_col_indices;
    float* dev_values;

    // allocate memory on device
    hipMalloc(&dev_row_indices, nnz * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));
    hipMalloc(&dev_values,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_indices, row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);

    // create COO matrix
    hipsparseCreateCoo(&sparse_matrix, rows, columns, nnz, dev_row_indices, dev_col_indices, dev_values, HIPSPARSE_INDEX_32I, 
                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    // transpose 

    // copy back


    // destroy matrix
    hipsparseDestroySpMat(sparse_matrix);

    // free device memory
    hipFree(dev_row_indices);
    hipFree(dev_col_indices);
    hipFree(dev_values);
}


int main(int argc, char* argv[]){
    cout << "Matrix 0\n";
    transpose_cuSparse_CSR("test_matrices/csr/0-test_csr.csv");
    cout << "Matrix 1\n";
    transpose_cuSparse_CSR("test_matrices/csr/1-bp_200_csr.csv");
    cout << "Matrix 2\n";
    transpose_cuSparse_CSR("test_matrices/csr/2-fs_183-1_csr.csv");
    cout << "Matrix 3\n";
    transpose_cuSparse_CSR("test_matrices/csr/3-fs_541_1_csr.csv");
    cout << "Matrix 4\n";
    transpose_cuSparse_CSR("test_matrices/csr/4-pores_2_csr.csv");
    cout << "Matrix 5\n";
    transpose_cuSparse_CSR("test_matrices/csr/5-shl_200_csr.csv");
    cout << "Matrix 6\n";
    transpose_cuSparse_CSR("test_matrices/csr/6-GD96_a_csr.csv");
    cout << "Matrix 7\n";
    transpose_cuSparse_CSR("test_matrices/csr/7-GD00_c_csr.csv");
    cout << "Matrix 8\n";
    transpose_cuSparse_CSR("test_matrices/csr/8-ch5-5-b3_csr.csv");
    cout << "Matrix 9\n";
    transpose_cuSparse_CSR("test_matrices/csr/9-dw256A_csr.csv");
    cout << "Matrix 10\n";
    transpose_cuSparse_CSR("test_matrices/csr/10-qh768_csr.csv");
    
    return 0;
}

