#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "../include/import_sparse_matrix.h"

using namespace std;

#define NUM_REPS 10



__global__ void warm_up_gpu(){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + tid; 
}


__global__ void transpose_COO(int* row_indices, int* column_indices, int nnz){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tmp;

    while (idx < nnz){
        // swap row and columns
        //printf("%d: %d, %d\n", idx, row_indices[idx], column_indices[idx]);
        tmp = row_indices[idx];
        row_indices[idx] = column_indices[idx];
        column_indices[idx] = tmp;
        //printf("%d: %d, %d\n", idx, row_indices[idx], column_indices[idx]);
        
        idx += gridDim.x * blockDim.x;
    }
}


__global__ void CSR2COO(int* row_offsets, int* row_indices, int rows){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int num_elements_in_row;

    while (idx < rows){
        // recreate row indices from offset
        num_elements_in_row = row_offsets[idx + 1] - row_offsets[idx];

        for (int i=0; i<num_elements_in_row; i++){
            row_indices[row_offsets[idx] + i] = idx;
        }

        idx += blockDim.x;
    }
}

__global__ void CSR2CSC(int rows, int columns, int nnz, int* num_elements_in_col, 
                        int* row_offsets_csr, int* column_indices_csr, float* values_csr,
                        int* row_indices_csc, int* column_offsets_csc,  float* values_csc, 
                        int* values_stored_from_col){
    int original_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = original_idx;

    // count number of non-zero elements per column
    while (idx < nnz){
        num_elements_in_col[column_indices_csr[idx]] += 1;
        idx += blockDim.x;
    }

    __syncthreads();

    // sum up the values to find column offsets
    idx = original_idx;
    while (idx < columns){
        for (int i=0; i<idx+1; i++){
            column_offsets_csc[idx + 1] += num_elements_in_col[i];
        }
        idx += blockDim.x;
    }

    __syncthreads();

    // insert row indices and values in the correct order
    idx = original_idx;
    int num_values;
    int col;
    
    while (idx < rows){
        num_values = row_offsets_csr[idx+1] - row_offsets_csr[idx];
        for (int i=0; i<num_values; i++){
            col = column_indices_csr[row_offsets_csr[idx] + i];
            row_indices_csc[column_offsets_csc[col] + values_stored_from_col[col]] = idx;
            values_csc[column_offsets_csc[col] + values_stored_from_col[col]] = values_csr[row_offsets_csr[idx] + i];
            values_stored_from_col[col] += 1;
        }
        idx += blockDim.x;
    }
}


void transpose_own_CSR(string file, string output_file){
    // load CSR matrix from file
    int rows, columns, nnz;
    int *row_offsets, *col_indices;
    float* values;

    csr_from_file(file, rows, columns, nnz, row_offsets, col_indices, values);

    // create arrays on device
    int *dev_row_offsets_csr, *dev_col_indices_csr;
    float* dev_values_csr;

    // allocate memory on device
    hipMalloc(&dev_row_offsets_csr, (rows+1) * sizeof(int));
    hipMalloc(&dev_col_indices_csr, nnz * sizeof(int));
    hipMalloc(&dev_values_csr,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_offsets_csr, row_offsets, (rows+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices_csr, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values_csr, values, nnz * sizeof(float), hipMemcpyHostToDevice);

    // create necessary buffer arrays and copy to device
    int *num_elements_in_col = (int*) malloc(columns * sizeof(int));
    int *values_stored_from_col = (int*) malloc(columns * sizeof(int));
    int *column_offsets_csc = (int*) malloc((columns + 1) * sizeof(int));

    for (int i=0; i<columns; i++){
        num_elements_in_col[i] = 0;
        values_stored_from_col[i] = 0;
        column_offsets_csc[i] = 0;
    }
    column_offsets_csc[columns] = 0;

    // allocate memory on device
    int *dev_num_elements_in_col, *dev_values_stored_from_col, *dev_column_offsets_csc, *dev_row_indices_csc;
    float* dev_values_csc;

    hipMalloc(&dev_num_elements_in_col, columns * sizeof(int));
    hipMalloc(&dev_values_stored_from_col, columns * sizeof(int));
    hipMalloc(&dev_column_offsets_csc, (columns+1) * sizeof(int));
    hipMalloc(&dev_row_indices_csc, nnz * sizeof(int));
    hipMalloc(&dev_values_csc, nnz * sizeof(float));

    // copy
    hipMemcpy(dev_num_elements_in_col, num_elements_in_col, columns * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values_stored_from_col, values_stored_from_col, columns * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_column_offsets_csc, column_offsets_csc, (columns+1) * sizeof(int), hipMemcpyHostToDevice);

    // create blocks and threads
    dim3 nBlocks(1, 1, 1);
    dim3 nThreads(1024, 1, 1);

    // call kernel
    CSR2CSC<<<nBlocks, nThreads>>>(rows, columns, nnz, dev_num_elements_in_col, dev_row_offsets_csr, 
                                    dev_col_indices_csr, dev_values_csr, dev_row_indices_csc, dev_column_offsets_csc,
                                    dev_values_csc, dev_values_stored_from_col);
    
    // synchronize
    hipDeviceSynchronize();

    // copy results back to host
    int *row_offsets_tp = (int*) malloc((columns+1) * sizeof(int));
    int *col_indices_tp = (int*) malloc(nnz * sizeof(int));
    float* values_tp = (float*) malloc(nnz * sizeof(float));

    hipMemcpy(col_indices_tp, dev_row_indices_csc, nnz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(row_offsets_tp, dev_column_offsets_csc, (columns + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(values_tp, dev_values_csc, nnz * sizeof(float), hipMemcpyDeviceToHost);

    // save transposed matrix to file
    transposed_csr_to_file(file, columns, rows, nnz, row_offsets_tp, col_indices_tp, values_tp);

    // free device memory
    hipFree(dev_row_offsets_csr);
    hipFree(dev_col_indices_csr);
    hipFree(dev_values_csr);

    hipFree(dev_num_elements_in_col);
    hipFree(dev_values_stored_from_col);
    hipFree(dev_column_offsets_csc);
    hipFree(dev_row_indices_csc);
    hipFree(dev_values_csc);

    // free host memory
    free(row_offsets);
    free(col_indices);
    free(values);
    free(num_elements_in_col);
    free(values_stored_from_col);
    free(column_offsets_csc);    
    free(row_offsets_tp);
    free(col_indices_tp);
    free(values_tp);
}



void transpose_own_COO(string file, string timing_file){
    // file to save execution time for bandwidth analysis
    std::ofstream myfile;
	myfile.open(timing_file);

    // load COO from file
    int rows, columns, nnz;
    int *row_indices, *col_indices;
    float* values;

    coo_from_file(file, rows, columns, nnz, row_indices, col_indices, values);

    // create arrays on device
    int *dev_row_indices, *dev_col_indices;

    // allocate memory on device
    hipMalloc(&dev_row_indices, nnz * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));

    // copy entries to device
    hipMemcpy(dev_row_indices, row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);

    // Create CUDA events to use for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // try different grid and block sizes and find fastest
    float min_time = INFINITY;
    int min_blocks;
    int min_threads;
    int possible_blocks = ceil(nnz / 1024);
    float milliseconds;
    for (int i=1; i<=possible_blocks; i++){
        dim3 nBlocks(i, 1, 1);

        if (i == 1){
            // test diferent numbers of threads
            for (int j=2; j<=1024; j*=2){
                dim3 nThreads(j, 1, 1);

                // start CUDA timer 
                hipEventRecord(start, 0);

                // invoke kernel NUM_REPS times 
                for (int k=0; k<NUM_REPS; k++){
                    transpose_COO<<<nBlocks, nThreads>>>(dev_row_indices, dev_col_indices, nnz);
                }

                // synchronize
                hipDeviceSynchronize();

                // stop CUDA timer
                hipEventRecord(stop, 0);
                hipEventSynchronize(stop); 

                // Calculate elapsed time
                milliseconds = 0;
                hipEventElapsedTime(&milliseconds, start, stop);

                // divide by NUM_REPS to get mean
                milliseconds /= NUM_REPS;

                if (milliseconds < min_time){
                    min_time = milliseconds;
                    min_blocks = i;
                    min_threads = j;
                }
            }
        }
        else {
            dim3 nThreads(1024, 1, 1);

            // start CUDA timer 
            hipEventRecord(start, 0);

            // invoke kernel NUM_REPS times 
            for (int k=0; k<NUM_REPS; k++){
                transpose_COO<<<nBlocks, nThreads>>>(dev_row_indices, dev_col_indices, nnz);
            }

            // synchronize
            hipDeviceSynchronize();

            // stop CUDA timer
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop); 

            // Calculate elapsed time
            milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);

            // divide by NUM_REPS to get mean
            milliseconds /= NUM_REPS;

            if (milliseconds < min_time){
                min_time = milliseconds;
                min_blocks = i;
                min_threads = 1024;
            }
        }
    }
    // find best configuration
    printf("Best configuration: %d, %d\n", min_blocks, min_threads);

    // save execution time and configuration to file
    myfile << milliseconds << "\n";
    myfile << rows << "\n";
    myfile << columns << "\n";
    myfile << nnz << "\n";
    myfile << min_blocks << "\n";
    myfile << min_threads << "\n";

    // copy back
    hipMemcpy(row_indices, dev_row_indices, nnz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(col_indices, dev_col_indices, nnz * sizeof(int), hipMemcpyDeviceToHost);

    printf("%d, %d\n", row_indices[1], col_indices[1]);

    // save result to file
    transposed_coo_to_file(file, columns, rows, nnz, row_indices, col_indices, values);

    // close file
	myfile.close();

    // free device memory
    hipFree(dev_row_indices);
    hipFree(dev_col_indices);

    // free host memory
    free(row_indices);
    free(col_indices);
    free(values);

    // free timer events
    hipEventDestroy(start);
    hipEventDestroy(stop);
}


void transpose_cuSparse_CSR(string file, string timing_file){
    // file to save execution time for bandwidth analysis
    std::ofstream myfile;
	myfile.open(timing_file);

    // load CSR matrix from file
    int rows, columns, nnz;
    int *row_offsets, *col_indices;
    float* values;

    csr_from_file(file, rows, columns, nnz, row_offsets, col_indices, values);

    // create arrays on device
    int *dev_row_offsets, *dev_col_indices;
    float* dev_values;

    // allocate memory on device
    hipMalloc(&dev_row_offsets, (rows+1) * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));
    hipMalloc(&dev_values,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_offsets, row_offsets, (rows+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);

    // Create CUDA events to use for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // reserve buffer space necessary for the transpose
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    int *dev_tp_row_indices, *dev_tp_col_offsets;
    float* dev_tp_values;
    hipMalloc(&dev_tp_row_indices, nnz * sizeof(int));
    hipMalloc(&dev_tp_col_offsets, (columns+1) * sizeof(int));
    hipMalloc(&dev_tp_values, nnz * sizeof(float));

    size_t buffer_size;
    hipsparseCsr2cscEx2_bufferSize(handle, rows, columns, nnz, dev_values, dev_row_offsets, dev_col_indices, 
                                    dev_tp_values, dev_tp_col_offsets, dev_tp_row_indices, HIP_R_32F, 
                                    HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
                                    &buffer_size); 

    void* buffer;
    hipMalloc(&buffer, buffer_size);

    // start CUDA timer 
    hipEventRecord(start, 0);

    // run NUM_REPS times
    for (int i=0; i<NUM_REPS; i++){
        // transpose by converting from CSR to CSC
        hipsparseCsr2cscEx2(handle, rows, columns, nnz, dev_values, dev_row_offsets, dev_col_indices, dev_tp_values, 
                            dev_tp_col_offsets, dev_tp_row_indices, HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, 
                            HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, buffer);
    }

    // stop CUDA timer
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop); 

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // divide by NUM_REPS to get mean
    milliseconds /= NUM_REPS;

    // save execution time and buffer size to file
    myfile << milliseconds << "\n";
    myfile << rows << "\n";
    myfile << columns << "\n";
    myfile << nnz << "\n";
    myfile << buffer_size << "\n";

    // copy results back to host
    int *row_offsets_tp = (int*) malloc((columns+1) * sizeof(int));
    int *col_indices_tp = (int*) malloc(nnz * sizeof(int));
    float* values_tp = (float*) malloc(nnz * sizeof(float));

    hipMemcpy(col_indices_tp, dev_tp_row_indices, nnz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(row_offsets_tp, dev_tp_col_offsets, (columns + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(values_tp, dev_tp_values, nnz * sizeof(float), hipMemcpyDeviceToHost);

    // save transposed matrix to file
    transposed_csr_to_file(file, columns, rows, nnz, row_offsets_tp, col_indices_tp, values_tp);

    // close file
	myfile.close();

    // free timer events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // destroy handle
    hipsparseDestroy(handle);

    // free device memory
    hipFree(dev_row_offsets);
    hipFree(dev_col_indices);
    hipFree(dev_values);

    hipFree(buffer);

    hipFree(dev_tp_row_indices);
    hipFree(dev_tp_col_offsets);
    hipFree(dev_tp_values);

    // free host memory
    free(row_offsets);
    free(col_indices);
    free(values);
    free(row_offsets_tp);
    free(col_indices_tp);
    free(values_tp);
}


void transpose_cuSparse_COO(string file){
    // load COO matrix from file
    int rows, columns, nnz;
    int *row_indices, *col_indices;
    float* values;

    coo_from_file(file, rows, columns, nnz, row_indices, col_indices, values);

    // create COO CUDA matrix using cuSparse 
    hipsparseSpMatDescr_t sparse_matrix;

    // void* for the three arrays
    int *dev_row_indices, *dev_col_indices;
    float* dev_values;

    // allocate memory on device
    hipMalloc(&dev_row_indices, nnz * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));
    hipMalloc(&dev_values,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_indices, row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);

    // create COO matrix
    hipsparseCreateCoo(&sparse_matrix, rows, columns, nnz, dev_row_indices, dev_col_indices, dev_values, HIPSPARSE_INDEX_32I, 
                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    // reserve necessary buffer space
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    const int alpha = 1;
    const int beta = 0;
    hipsparseDnMatDescr_t dense_matrix;
    float* dev_dmat_values;
    hipMalloc(&dev_dmat_values, rows * columns * sizeof(float));
    hipsparseCreateDnMat(&dense_matrix, rows, columns, rows, dev_dmat_values, HIP_R_32F, HIPSPARSE_ORDER_ROW);

    size_t buffer_size;
    hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, 
                            sparse_matrix, dense_matrix, &beta, dense_matrix, HIP_R_32F, HIPSPARSE_SPMM_COO_ALG1, 
                            &buffer_size);
    void* buffer;
    hipMalloc(&buffer, buffer_size);

    // transpose
    hipsparseSpMM(handle, HIPSPARSE_OPERATION_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, sparse_matrix, 
                    dense_matrix, &beta, dense_matrix, HIP_R_32F, HIPSPARSE_SPMM_COO_ALG1, buffer);

    // save values back into sparse matrix
    hipsparseDenseToSparse_bufferSize(handle, dense_matrix, sparse_matrix, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &buffer_size);
    void* buffer_convert;
    hipMalloc(&buffer_convert, buffer_size);
    hipsparseDenseToSparse_analysis(handle, dense_matrix, sparse_matrix, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, buffer_convert);
    hipsparseDenseToSparse_convert(handle, dense_matrix, sparse_matrix, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, buffer_convert);

    // copy back to host
    hipsparseIndexType_t index_type = HIPSPARSE_INDEX_32I;
    hipsparseIndexBase_t index_base = HIPSPARSE_INDEX_BASE_ZERO;
    hipDataType data_type = HIP_R_32F;
    hipsparseCooGet(sparse_matrix, (int64_t*)&rows, (int64_t*)&columns, (int64_t*)&nnz, 
                    (void **)&dev_row_indices, (void **)&dev_col_indices, (void **)&dev_values, 
                    &index_type, &index_base, &data_type);

    hipMemcpy(row_indices, dev_row_indices, nnz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(col_indices, dev_col_indices, nnz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(values, dev_values, nnz * sizeof(float), hipMemcpyDeviceToHost);

    for (int i=0; i<nnz; i++){
        printf("%f\n", values);
    }
               
    // write transposed matrix to file
    transposed_coo_to_file(file, columns, rows, nnz, row_indices, col_indices, values);                            

    // destroy matrix
    hipsparseDestroySpMat(sparse_matrix);
    hipsparseDestroyDnMat(dense_matrix);

    // destroy handle
    hipsparseDestroy(handle);

    // free device memory
    hipFree(dev_row_indices);
    hipFree(dev_col_indices);
    hipFree(dev_values);
    hipFree(buffer);
    hipFree(dev_dmat_values);
    hipFree(buffer_convert);

    // free host memory 
    free(row_indices);
    free(col_indices);
    free(values);
}


int main(int argc, char* argv[]){
    if (argc < 2){
        throw runtime_error("Please choose a strategy");
    }

    // Strategy 0: cuSPARSE CSR
    if (atoi(argv[1]) == 0){
        printf("Use CSR format and the cuSPARSE library.\n");

        // check which test matrix to use
        if (argc < 3){
            throw runtime_error("Please choose a test matrix");
        }

        string argv2 = argv[2];
        if (argv2 == "all"){
            for (int i=1; i<11; i++){
                printf("Transposing matrix %d\n", i);
                transpose_cuSparse_CSR("test_matrices/csr/" + to_string(i) + "_csr.csv", 
                                        "output/csr_cusparse_" + to_string(i) + ".csv");
            }
        }
        else {
            printf("Transposing matrix %d\n", atoi(argv[2]));
            transpose_cuSparse_CSR("test_matrices/csr/" + to_string(atoi(argv[2])) + "_csr.csv",
                                    "output/csr_cusparse_" + to_string(atoi(argv[2])) + ".csv");
        }
    }

    // Strategy 1: cuSPARSE COO - Currently not working
    if (atoi(argv[1]) == 1){
        printf("Use COO format and the cuSPARSE library.\n");

        // check which test matrix to use
        if (argc < 3){
            throw runtime_error("Please choose a test matrix");
        }

        string argv2 = argv[2];
        if (argv2 == "all"){
            for (int i=1; i<11; i++){
                printf("Transposing matrix %d\n", i);
                transpose_cuSparse_COO("test_matrices/coo/" + to_string(i) + "_coo.csv");
            }
        }
        else {
            printf("Transposing matrix %d\n", atoi(argv[2]));
            transpose_cuSparse_COO("test_matrices/coo/" + to_string(atoi(argv[2])) + "_coo.csv");
        }
    }

    // Strategy 2: own COO transpose kernel
    if (atoi(argv[1]) == 2){
        printf("Use COO format and own kernel.\n");

        // check which test matrix to use
        if (argc < 3){
            throw runtime_error("Please choose a test matrix");
        }

        string argv2 = argv[2];
        if (argv2 == "all"){
            for (int i=1; i<11; i++){
                printf("Transposing matrix %d\n", i);
                transpose_own_COO("test_matrices/coo/" + to_string(i) + "_coo.csv",
                                    "output/coo_own_" + to_string(atoi(argv[2])) + ".csv");
            }
        }
        else {
            printf("Transposing matrix %d\n", atoi(argv[2]));
            transpose_own_COO("test_matrices/coo/" + to_string(atoi(argv[2])) + "_coo.csv",
                                "output/coo_own_" + to_string(atoi(argv[2])) + ".csv");
        }
        
    }

    // Strategy 3: own CSR2CSC kernel
    if (atoi(argv[1]) == 3){
        printf("Use CSR format and own CSR2CSC kernel.\n");

        // check which test matrix to use
        if (argc < 3){
            throw runtime_error("Please choose a test matrix");
        }

        string argv2 = argv[2];
        if (argv2 == "all"){
            for (int i=1; i<11; i++){
                printf("Transposing matrix %d\n", i);
                transpose_own_CSR("test_matrices/csr/" + to_string(i) + "_csr.csv",
                                    "output/csr_own_" + to_string(atoi(argv[2])) + ".csv");
            }
        }
        else {
            printf("Transposing matrix %d\n", atoi(argv[2]));
            transpose_own_COO("test_matrices/csr/" + to_string(atoi(argv[2])) + "_csr.csv",
                                "output/csr_own_" + to_string(atoi(argv[2])) + ".csv");
        }
        
    }


    
    return 0;
}

