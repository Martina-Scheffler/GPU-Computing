#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "../include/import_sparse_matrix.h"

using namespace std;

#define NUM_REPS 10



__global__ void warm_up_gpu(){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + tid; 
}


void transpose_cuSparse_CSR(string file){
    // load CSR matrix from file
    int rows, columns, nnz;
    int *row_offsets, *col_indices;
    float* values;

    csr_from_file(file, rows, columns, nnz, row_offsets, col_indices, values);

    // create CSR matrix using cuSparse
    hipsparseSpMatDescr_t sparse_matrix;

    // void* for the three arrays
    int *dev_row_offsets, *dev_col_indices;
    float* dev_values;

    // allocate memory on device
    hipMalloc(&dev_row_offsets, (rows+1) * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));
    hipMalloc(&dev_values,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_offsets, row_offsets, (rows+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);
    
    // create CSR matrix
    hipsparseCreateCsr(&sparse_matrix, rows, columns, nnz, dev_row_offsets, dev_col_indices, dev_values, 
                        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    // reserve buffer space necessary for the transpose
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    int *dev_tp_row_indices, *dev_tp_col_offsets;
    float* dev_tp_values;
    hipMalloc(&dev_tp_row_indices, nnz * sizeof(int));
    hipMalloc(&dev_tp_col_offsets, (columns + 1) * sizeof(int));
    hipMalloc(&dev_tp_values, nnz * sizeof(float));

    size_t buffer_size;

    hipsparseCsr2cscEx2_bufferSize(handle, rows, columns, nnz, dev_values, dev_row_offsets, dev_col_indices, 
                                    dev_tp_values, dev_tp_col_offsets, dev_tp_row_indices, HIP_R_32F, 
                                    HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG_DEFAULT,
                                    &buffer_size); 

    cout << buffer_size << "\n"; 
                              
                              
    // transpose by converting from CSR to CSC
    void* buffer;
    hipMalloc(&buffer, buffer_size);
    hipsparseCsr2cscEx2(handle, row, columns, nnz, dev_values, dev_row_offsets, dev_col_indices, dev_tp_values, 
                        dev_tp_col_offsets, dev_tp_row_indices, HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, 
                        HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG_DEFAULT, buffer);

    // copy results back to host
    int *row_indices = (int*) malloc(nnz * sizeof(int));
    int *col_offsets = (int*) malloc((columns + 1) * sizeof(int));
    float* values_csc = (float*) malloc(nnz * sizeof(float));

    hipMemcpy(row_indices, dev_tp_row_indices, nnz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(col_offsets, dev_tp_col_offsets, (columns + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(values_csc, dev_tp_values, nnz * sizeof(float), hipMemcpyDeviceToHost);
                   
    
    // destroy matrix
    hipsparseDestroySpMat(sparse_matrix);

    // destroy handle
    hipsparseDestroy(handle);

    // free device memory
    hipFree(dev_row_offsets);
    hipFree(dev_col_indices);
    hipFree(dev_values);

    hipFree(dev_tp_row_indices);
    hipFree(dev_tp_col_offsets);
    hipFree(dev_tp_values);
}


void transpose_cuSparse_COO(string file){
    cout << "Running cuSparse COO transpose" << "\n";
    // load COO matrix from file
    int rows, columns, nnz;
    int *row_indices, *col_indices;
    float* values;

    coo_from_file(file, rows, columns, nnz, row_indices, col_indices, values);

    // create COO CUDA matrix using cuSparse 
    hipsparseSpMatDescr_t sparse_matrix;

    // void* for the three arrays
    int *dev_row_indices, *dev_col_indices;
    float* dev_values;

    // allocate memory on device
    hipMalloc(&dev_row_indices, nnz * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));
    hipMalloc(&dev_values,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_indices, row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);

    // create COO matrix
    hipsparseCreateCoo(&sparse_matrix, rows, columns, nnz, dev_row_indices, dev_col_indices, dev_values, HIPSPARSE_INDEX_32I, 
                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    // transpose 

    // copy back


    // destroy matrix
    hipsparseDestroySpMat(sparse_matrix);

    // free device memory
    hipFree(dev_row_indices);
    hipFree(dev_col_indices);
    hipFree(dev_values);
}


int main(int argc, char* argv[]){
    transpose_cuSparse_CSR("test_matrices/coo/1-bp_200_coo.csv");
    
    return 0;
}

