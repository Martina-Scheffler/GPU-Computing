#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "../include/import_sparse_matrix.h"

using namespace std;

#define NUM_REPS 10



__global__ void warm_up_gpu(){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + tid; 
}


void transpose_cuSparse_CSR(string file){
    // load CSR matrix from file
    int rows, columns, nnz;
    int *row_offsets, *col_indices;
    float* values;

    csr_from_file(file, rows, columns, nnz, row_offsets, col_indices, values);

    // create arrays on device
    int *dev_row_offsets, *dev_col_indices;
    float* dev_values;

    // allocate memory on device
    hipMalloc(&dev_row_offsets, (rows+1) * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));
    hipMalloc(&dev_values,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_offsets, row_offsets, (rows+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);

    // reserve buffer space necessary for the transpose
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    int *dev_tp_row_indices, *dev_tp_col_offsets;
    float* dev_tp_values;
    hipMalloc(&dev_tp_row_indices, nnz * sizeof(int));
    hipMalloc(&dev_tp_col_offsets, (columns+1) * sizeof(int));
    hipMalloc(&dev_tp_values, nnz * sizeof(float));

    size_t buffer_size;

    hipsparseCsr2cscEx2_bufferSize(handle, rows, columns, nnz, dev_values, dev_row_offsets, dev_col_indices, 
                                    dev_tp_values, dev_tp_col_offsets, dev_tp_row_indices, HIP_R_32F, 
                                    HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
                                    &buffer_size); 
                              
    // transpose by converting from CSR to CSC
    void* buffer;
    hipMalloc(&buffer, buffer_size);
    hipsparseCsr2cscEx2(handle, rows, columns, nnz, dev_values, dev_row_offsets, dev_col_indices, dev_tp_values, 
                        dev_tp_col_offsets, dev_tp_row_indices, HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, 
                        HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, buffer);

    // copy results back to host
    int *row_offsets_tp = (int*) malloc((columns+1) * sizeof(int));
    int *col_indices_tp = (int*) malloc(nnz * sizeof(int));
    float* values_tp = (float*) malloc(nnz * sizeof(float));

    hipMemcpy(col_indices_tp, dev_tp_row_indices, nnz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(row_offsets_tp, dev_tp_col_offsets, (columns + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(values_tp, dev_tp_values, nnz * sizeof(float), hipMemcpyDeviceToHost);

    // save transposed matrix to file
    transposed_csr_to_file(file, columns, rows, nnz, row_offsets_tp, col_indices_tp, values_tp);

    // destroy handle
    hipsparseDestroy(handle);

    // free device memory
    hipFree(dev_row_offsets);
    hipFree(dev_col_indices);
    hipFree(dev_values);

    hipFree(dev_tp_row_indices);
    hipFree(dev_tp_col_offsets);
    hipFree(dev_tp_values);

    // free host memory
    free(row_offsets);
    free(col_indices);
    free(values);
    free(row_offsets_tp);
    free(col_indices_tp);
    free(values_tp);
}


void transpose_cuSparse_COO(string file){
    // load COO matrix from file
    int rows, columns, nnz;
    int *row_indices, *col_indices;
    float* values;

    coo_from_file(file, rows, columns, nnz, row_indices, col_indices, values);

    // create COO CUDA matrix using cuSparse 
    hipsparseSpMatDescr_t sparse_matrix;

    // void* for the three arrays
    int *dev_row_indices, *dev_col_indices;
    float* dev_values;

    // allocate memory on device
    hipMalloc(&dev_row_indices, nnz * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));
    hipMalloc(&dev_values,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_indices, row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);

    // create COO matrix
    hipsparseCreateCoo(&sparse_matrix, rows, columns, nnz, dev_row_indices, dev_col_indices, dev_values, HIPSPARSE_INDEX_32I, 
                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    // transpose 

    // copy back


    // destroy matrix
    hipsparseDestroySpMat(sparse_matrix);

    // free device memory
    hipFree(dev_row_indices);
    hipFree(dev_col_indices);
    hipFree(dev_values);

    // free host memory 
    free(row_indices);
    free(col_indices);
    free(values);
}


int main(int argc, char* argv[]){
    if (argc < 2){
        throw runtime_error("Please choose a strategy");
    }
    if (atoi(argv[1]) == 0){
        printf("Use CSR format and the cuSPARSE library.\n");

        // check which test matrix to use
        if (argc < 3){
            throw runtime_error("Please choose a test matrix");
        }

        if (to_string(argv[2]) == "all"){
            for (int i=1; i<11; i++){
                printf("Transposing matrix %d\n", i);
                transpose_cuSparse_CSR("test_matrices/csr/" + i + "_csr.csv");
            }
        }
        else {
            printf("Transposing matrix %d\n", atoi(argv[2]));
            transpose_cuSparse_CSR("test_matrices/csr/" + atoi(argv[2]) + "_csr.csv");
        }
    }
    
    return 0;
}

