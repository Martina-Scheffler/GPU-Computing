#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include "../include/import_sparse_matrix.h"

using namespace std;

#define NUM_REPS 10



__global__ void warm_up_gpu(){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float ia, ib;
    ia = ib = 0.0f;
    ib += ia + tid; 
}


void transpose_cuSparse_CSR(string file){
    // load CSR matrix from file
    int rows, columns, nnz;
    int *row_offsets, *col_indices;
    float* values;

    csr_from_file(file, rows, columns, nnz, row_offsets, col_indices, values);

    // for (int i=0; i<rows+1; i++){
    //     printf("%d\n", row_offsets[i]);
    // }

    // create CSR matrix using cuSparse
    hipsparseSpMatDescr_t sparse_matrix;

    // create arrays on device
    int *dev_row_offsets, *dev_col_indices;
    float* dev_values;

    // allocate memory on device
    hipMalloc(&dev_row_offsets, (rows+1) * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));
    hipMalloc(&dev_values,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_offsets, row_offsets, (rows+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);
    
    // create CSR matrix
    // hipsparseCreateCsr(&sparse_matrix, rows, columns, nnz, dev_row_offsets, dev_col_indices, dev_values, 
    //                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

    // reserve buffer space necessary for the transpose
    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    int *dev_tp_row_indices, *dev_tp_col_offsets;
    float* dev_tp_values;
    hipMalloc(&dev_tp_row_indices, nnz * sizeof(int));
    hipMalloc(&dev_tp_col_offsets, (columns+1) * sizeof(int));
    hipMalloc(&dev_tp_values, nnz * sizeof(float));

    size_t buffer_size;

    hipsparseCsr2cscEx2_bufferSize(handle, rows, columns, nnz, dev_values, dev_row_offsets, dev_col_indices, 
                                    dev_tp_values, dev_tp_col_offsets, dev_tp_row_indices, HIP_R_32F, 
                                    HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
                                    &buffer_size); 

    cout << buffer_size << "\n"; 
                              
    // transpose by converting from CSR to CSC
    void* buffer;
    hipMalloc(&buffer, buffer_size);
    hipsparseStatus_t result = hipsparseCsr2cscEx2(handle, rows, columns, nnz, dev_values, dev_row_offsets, dev_col_indices, dev_tp_values, 
                        dev_tp_col_offsets, dev_tp_row_indices, HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, 
                        HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, buffer);

    cout << result << "\n";

    // copy results back to host
    int *row_offsets_tp = (int*) malloc((columns+1) * sizeof(int));
    int *col_indices_tp = (int*) malloc(nnz * sizeof(int));
    float* values_tp = (float*) malloc(nnz * sizeof(float));

    hipMemcpy(col_indices_tp, dev_tp_row_indices, nnz * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(row_offsets_tp, dev_tp_col_offsets, (columns + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(values_tp, dev_tp_values, nnz * sizeof(float), hipMemcpyDeviceToHost);

    for (int i=0; i<columns+1; i++){
        printf("%d\n", row_offsets_tp[i]);
    }

    // save transposed matrix to file
    transposed_csr_to_file(file, columns, rows, nnz, row_offsets_tp, col_indices_tp, values_tp);
    
    // // destroy matrix
    // hipsparseDestroySpMat(sparse_matrix);

    // destroy handle
    hipsparseDestroy(handle);

    // free device memory
    hipFree(dev_row_offsets);
    hipFree(dev_col_indices);
    hipFree(dev_values);

    hipFree(dev_tp_row_indices);
    hipFree(dev_tp_col_offsets);
    hipFree(dev_tp_values);
}


void transpose_cuSparse_COO(string file){
    cout << "Running cuSparse COO transpose" << "\n";
    // load COO matrix from file
    int rows, columns, nnz;
    int *row_indices, *col_indices;
    float* values;

    coo_from_file(file, rows, columns, nnz, row_indices, col_indices, values);

    // create COO CUDA matrix using cuSparse 
    hipsparseSpMatDescr_t sparse_matrix;

    // void* for the three arrays
    int *dev_row_indices, *dev_col_indices;
    float* dev_values;

    // allocate memory on device
    hipMalloc(&dev_row_indices, nnz * sizeof(int));
    hipMalloc(&dev_col_indices, nnz * sizeof(int));
    hipMalloc(&dev_values,  nnz * sizeof(float));

    // copy entries to device
    hipMemcpy(dev_row_indices, row_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_indices, col_indices, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_values, values, nnz * sizeof(float), hipMemcpyHostToDevice);

    // create COO matrix
    hipsparseCreateCoo(&sparse_matrix, rows, columns, nnz, dev_row_indices, dev_col_indices, dev_values, HIPSPARSE_INDEX_32I, 
                        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    // transpose 

    // copy back


    // destroy matrix
    hipsparseDestroySpMat(sparse_matrix);

    // free device memory
    hipFree(dev_row_indices);
    hipFree(dev_col_indices);
    hipFree(dev_values);
}


int main(int argc, char* argv[]){
    transpose_cuSparse_CSR("test_matrices/csr/2-fs_183_1_csr.csv");
    
    return 0;
}

