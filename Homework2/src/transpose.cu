#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>


#include "../include/matrix_generation.h"

using namespace std;

#define TILE_DIMENSION 4
#define BLOCK_ROWS 1

__global__ void transposeSimple(int* A, int* A_T){
	int x = blockIdx.x * TILE_DIMENSION + threadIdx.x;
	int y = blockIdx.y * TILE_DIMENSION + threadIdx.y;
    int width = gridDim.x * TILE_DIMENSION;

    for(int i=0; i<TILE_DIMENSION; i+=BLOCK_ROWS){
        A_T[x * width + (y + i)] = A[(y + i) * width + x];
    }
}

__global__ void transposeCoalesced(int *A, int *A_T){
    __shared__ int tile[TILE_DIMENSION][TILE_DIMENSION + 1];  // +1 in y to avoid bank conflicts

    int x = blockIdx.x * TILE_DIMENSION + threadIdx.x;
    int y = blockIdx.y * TILE_DIMENSION + threadIdx.y;
    int width = gridDim.x * TILE_DIMENSION;

    for (int i=0; i<TILE_DIMENSION; i+=BLOCK_ROWS){
        tile[threadIdx.y + i][threadIdx.x] = A[(y + i) * width + x];
    }
        
    __syncthreads();

    x = blockIdx.y * TILE_DIMENSION + threadIdx.x;  // transpose block offset
    y = blockIdx.x * TILE_DIMENSION + threadIdx.y;

    for (int i=0; i<TILE_DIMENSION; i+=BLOCK_ROWS){
        A_T[(y + i) * width + x] = tile[threadIdx.x][threadIdx.y + i];
    }
}

__global__ void transposeDiagonal(int *A, int *A_T){
    __shared__ int tile[TILE_DIMENSION][TILE_DIMENSION + 1];

    // diagonal reordering
    int blockIdx_y = blockIdx.x;
    int blockIdx_x = (blockIdx.x + blockIdx.y) % gridDim.x;

    int x = blockIdx_x * TILE_DIMENSION + threadIdx.x;
    int y = blockIdx_y * TILE_DIMENSION + threadIdx.y;
    int width = gridDim.x * TILE_DIMENSION;


    for (int i=0; i<TILE_DIMENSION; i+=BLOCK_ROWS){
        tile[threadIdx.y + i][threadIdx.x] = A[(y + i) * width + x];
    }
        
    __syncthreads();

    x = blockIdx_y * TILE_DIMENSION + threadIdx.x;  
    y = blockIdx_x * TILE_DIMENSION + threadIdx.y;

    for (int i=0; i<TILE_DIMENSION; i+=BLOCK_ROWS){
        A_T[(y + i) * width + x] = tile[threadIdx.x][threadIdx.y + i];
    }
}


int main(int argc, char* argv[]){
    // check if the matrix size was provided
	if (argc < 2){
		throw runtime_error("Please enter an integer N as argument to generate a matrix of size 2^N x 2^N.");
	}
    
    if (atoi(argv[1]) == 0){
        // use zero for something later
    }
    else {
        int size = pow(2, atoi(argv[1]));
        int N = size * size;

        cout << size << endl;

		// call matrix generation with command line argument and receive matrix back
		int* A = generate_continous_matrix(size);

        for (int i=0; i<size; i++){
            for (int j=0; j<size; i++){
                cout << i*size + j << "\t";
            }
            cout << "\n";
        }

        // allocate memory on host
        int* A_T = (int*) malloc(N * sizeof(int));

        // allocate memory on device
        int *dev_A, *dev_A_T;

        hipMalloc(&dev_A, N * sizeof(int));
        hipMalloc(&dev_A_T, N * sizeof(int));

        // copy matrix to device
        hipMemcpy(dev_A, A, N * sizeof(int), hipMemcpyHostToDevice);

        // start CUDA timer

        // determine kernel dimensions
        dim3 nBlocks (size / TILE_DIMENSION, size / TILE_DIMENSION, 1);
        dim3 nThreads (TILE_DIMENSION, BLOCK_ROWS, 1);

        // run kernel
        transposeSimple<<<nBlocks, nThreads>>>(dev_A, dev_A_T);
        // transposeCoalesced<<<nBlocks, nThreads>>>(A, A_T);
        // transposeDiagonal<<nBlocks, nThreads>>>(A, A_T)

        // synchronize
        hipDeviceSynchronize();

        // stop CUDA timer

        // copy back
        hipMemcpy(dev_A_T, A_T, N * sizeof(int), hipMemcpyDeviceToHost);

        // display result
        // for (int i=0; i<size; i++){
        //     for (int j=0; j<size; i++){
        //         cout << A_T[i*size + j] << "\t";
        //     }
        //     cout << "\n";
        // }
        
        // free memory on device
        hipFree(dev_A);
        hipFree(dev_A_T);

        // free memory on host
        free(A);
        free(A_T);

        return 0;
    }

}